#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

template <typename T>
__global__ void GeluKernel(const T* in, T* out, int n_elements)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int chunk_size = blockDim.x * gridDim.x;

    const T scale = sqrt(T(2) / HIP_PI);

    for (int i = gid; i < n_elements; i += chunk_size)
    {
        T x = in[i];
        T cdf = T(1) + tanh(scale * (x + T(0.044715)) * (x * x * x));
        cdf *= T(0.5);
        out[i] = x * cdf;
    }
}

void GeluKernelLauncher(const float* in, float* out, int n_elements, int n_dev, hipStream_t stream)
{
    int block_size = 1024;
    int n_block = 2 * n_dev;
    GeluKernel<<<n_block, block_size, 0, stream>>>(in, out, n_elements);
}